#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

__global__ void reduce(int *g_idata, int *g_odata)
{
	extern __shared__ int sdata[THREADS_PER_BLOCK];
	
	unsigned int tid = threadIdx.x;
 	unsigned int offset = tid * (SIZE / THREADS_PER_BLOCK);	

	sdata[tid] = 0;


	for(unsigned int k=0; k < SIZE/THREADS_PER_BLOCK; k++)
	{
		sdata[tid] += g_idata[offset+k];
	}

	__syncthreads();

	for(unsigned int s=1; s < blockDim.x; s*=2)
	{
		if(tid % (2*s) == 0)
			sdata[tid] += sdata[tid+s];
		__syncthreads();
	}

	if(tid == 0)
		*g_odata = sdata[0];
}


int main()
{
	int *h_idata, *h_odata;
	int *d_idata, *d_odata;
	clock_t start, end;

        h_idata = (int*) malloc(SIZE * sizeof(int));
	h_odata = (int*) malloc(sizeof(int));
	hipMalloc((void **) &d_idata, SIZE * sizeof(int));
	hipMalloc((void **) &d_odata, sizeof(int));


	for(int i=0; i<SIZE; i++)
	{
		scanf("%d", &h_idata[i]);
	}
	
	start = clock();	

 	hipMemcpy(d_idata, h_idata, SIZE * sizeof(int), hipMemcpyHostToDevice);
	
	dim3 dimBlock(THREADS_PER_BLOCK, 1, 1);
	dim3 dimGrid(1, 1, 1);

	reduce<<<dimGrid, dimBlock>>>(d_idata, d_odata);

	hipMemcpy(h_odata, d_odata, sizeof(int), hipMemcpyDeviceToHost);

	end = clock();

	printf("%d\n", *h_odata);	
	printf("%f\n", (double) (end - start));
	
	hipFree(d_idata);
	hipFree(d_odata);	

	return 0;

}
